#include "hip/hip_runtime.h"
#include "BMPLoader.h"

__global__ void kernelMakeBlackAndWhite(RGB* pixVec, RGB* pixVecNew, int imageWidth);

#define NOFTHREADS 128

int main(void)
{

	std::string filename[6];
	filename[0] = "flower.bmp";
	filename[1] = "dimix2.bmp";
	filename[2] = "balloon.bmp";
	filename[3] = "redgreen.bmp";
	filename[4] = "waterdrop.bmp";
	filename[5] = "redgreenSmall.bmp";

	std::string newFilename[6];
	newFilename[0] = "flowerNew";
	newFilename[1] = "dimix2New";
	newFilename[2] = "balloonNew";
	newFilename[3] = "redgreen";
	newFilename[4] = "waterdropNew";
	newFilename[5] = "redgreenSmallNew";

	RGB* readPixels;
	RGB* newPixels;
	RGB* dev_pixels;
	RGB* dev_pixels_new;
	BMPLoader b;
	int idx = 0;

	unsigned int imageSize = b.getImageSize((filename[idx]).c_str());

	//CPU Speicher allokieren
	readPixels = (RGB*)malloc(imageSize);
	newPixels = (RGB*)malloc(imageSize);


	//Bild importieren
	b.loadBMP((filename[idx]).c_str(), readPixels);

	//GPU Speicher allokieren
	hipMalloc((void**) &dev_pixels, imageSize);
	hipMalloc((void**) &dev_pixels_new, imageSize);

	//Pixel auf die GPU kopieren
	hipMemcpy(dev_pixels, readPixels, imageSize, hipMemcpyHostToDevice);

	int NofBlocks = b.bih.biHeight;
	int NofPixels = b.bih.biHeight*b.bih.biWidth;
	// Kernel launchen mit Anzahl Blocks = Hoehe des Bildes
	kernelMakeBlackAndWhite << <NofBlocks, NOFTHREADS >> >(dev_pixels, dev_pixels_new, NofPixels);

	//Berechnetes Bild zurueckkopieren
	hipMemcpy(newPixels, dev_pixels_new, imageSize, hipMemcpyDeviceToHost);

	//GPU Speicher freigeben
	hipFree(dev_pixels);
	hipFree(dev_pixels_new);

	//Bild schreiben (von CPU aus)
	b.writeBMP(newPixels, (newFilename[idx]).c_str());

	//CPU Speicher freigeben
	free(readPixels);
	free(newPixels);

	return 0;
}


// Filter fuer schwarz-weiss
__global__ void kernelMakeBlackAndWhite(RGB* pixVec, RGB* pixVecNew, int NofPixels){

	int x = threadIdx.x + blockIdx.x * blockDim.x;

	while (x < NofPixels)
	{
		float sum = pixVec[x].rgbRed + pixVec[x].rgbGreen + pixVec[x].rgbBlue;

		if (sum >= 3.0*255.0 / 2.0) {
			pixVecNew[x].rgbRed = 255;
			pixVecNew[x].rgbGreen = 255;
			pixVecNew[x].rgbBlue = 255;
		}
		else {
			pixVecNew[x].rgbRed = 0;
			pixVecNew[x].rgbGreen = 0;
			pixVecNew[x].rgbBlue = 0;
		}

		x += blockDim.x*gridDim.x;
	
	}

}